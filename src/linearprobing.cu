#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdint.h"
#include "vector"
#include "linearprobing.h"

// 32 bit Murmur3 hash
__device__ uint32_t hash(uint32_t k)
{
    k ^= k >> 16;
    k *= 0x85ebca6b;
    k ^= k >> 13;
    k *= 0xc2b2ae35;
    k ^= k >> 16;
    return k & (kHashTableCapacity-1);
}

// Create a hash table. For linear probing, this is just an array
// of KeyValues. The hash table is
KeyValue* create_hashtable(uint32_t capacity) 
{
    // Allocate memory
    KeyValue* hashtable;
    hipMalloc(&hashtable, sizeof(KeyValue) * kHashTableCapacity);

    // Initialize hash table to empty
    static_assert(kEmpty == 0xffffffff, "memset expected kEmpty=0xffffffff");
    hipMemset(hashtable, 0xff, sizeof(KeyValue) * kHashTableCapacity);

    return hashtable;
}

__global__ void gpu_hashtable_insert(KeyValue* hashtable, const KeyValue* kvs, unsigned int numkvs)
{
    unsigned int threadid = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadid < numkvs)
    {
        uint32_t key = kvs[threadid].key;
        uint32_t value = kvs[threadid].value;
        uint32_t slot = hash(key);

        while (true)
        {
            uint32_t prev = atomicCAS(&hashtable[slot].key, kEmpty, key);
            if (prev == kEmpty || prev == key)
            {
                hashtable[slot].value = value;
                break;
            }

            slot = (slot + 1) & (kHashTableCapacity-1);
        }
    }
}
 
void insert_hashtable(KeyValue* pHashTable, const KeyValue* kvs, uint32_t num_kvs)
{
    // Copy the keyvalues to the GPU
    KeyValue* device_kvs;
    hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
    hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

    // Have CUDA calculate the thread block size
    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

    // Create events for GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Insert all the keys into the hash table
    int gridsize = ((uint32_t)num_kvs + threadblocksize - 1) / threadblocksize;
    gpu_hashtable_insert<<<gridsize, threadblocksize>>>(pHashTable, device_kvs, (uint32_t)num_kvs);

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float seconds = milliseconds / 1000.0f;
    printf("    GPU inserted %d items in %f ms (%f million keys/second)\n", 
        num_kvs, milliseconds, num_kvs / (double)seconds / 1000000.0f);

    hipFree(device_kvs);
}

__global__ void gpu_iterate_hashtable(KeyValue* pHashTable, KeyValue* kvs, uint32_t* kvs_size) 
{
    unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadid < kHashTableCapacity) 
    {
        if (pHashTable[threadid].key != kEmpty) 
        {
            uint32_t size = atomicAdd(kvs_size, 1);
            kvs[size] = pHashTable[threadid];
        }
    }
}

std::vector<KeyValue> iterate_hashtable(KeyValue* pHashTable)
{
    uint32_t* device_num_kvs;
    hipMalloc(&device_num_kvs, sizeof(uint32_t));
    hipMemset(device_num_kvs, 0, sizeof(uint32_t));

    KeyValue* device_kvs;
    hipMalloc(&device_kvs, sizeof(KeyValue) * kNumKeyValues);

    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_iterate_hashtable, 0, 0);

    int gridsize = (kHashTableCapacity + threadblocksize - 1) / threadblocksize;
    gpu_iterate_hashtable<<<gridsize, threadblocksize>>>(pHashTable, device_kvs, device_num_kvs);

    uint32_t num_kvs;
    hipMemcpy(&num_kvs, device_num_kvs, sizeof(uint32_t), hipMemcpyDeviceToHost);

    std::vector<KeyValue> kvs;
    kvs.resize(num_kvs);

    hipMemcpy(kvs.data(), device_kvs, sizeof(KeyValue) * num_kvs, hipMemcpyDeviceToHost);

    hipFree(device_kvs);
    hipFree(device_num_kvs);

    return kvs;
}

void destroy_hashtable(KeyValue* pHashTable)
{
    hipFree(pHashTable);
}
